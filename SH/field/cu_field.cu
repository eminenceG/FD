#include"field.h"
#include"../gpu.h"
#include"../param/param.h"
#include<cstdio>

/* init fld in CUDA */
void FIELD::init_gpu_full(int deviceid,PARAM & param){
  nx=param.nx;
  nz=param.nz;
  int nz1=param.a_nz1[deviceid];
  int nz2=param.a_nz2[deviceid];
  int tnz=nz2-nz1+1+2*radius;
  safecall(hipMalloc((void**)&(V  ),sizeof(float)*nx*tnz));
  safecall(hipMalloc((void**)&(Tx ),sizeof(float)*nx*tnz));
  safecall(hipMalloc((void**)&(Tz ),sizeof(float)*nx*tnz));
  hipMemset(V  ,  0,sizeof(float)*nx*tnz); V    -=(nz1-radius)*nx;
  hipMemset(Tx ,  0,sizeof(float)*nx*tnz); Tx   -=(nz1-radius)*nx;
  hipMemset(Tz ,  0,sizeof(float)*nx*tnz); Tz   -=(nz1-radius)*nx;
}
