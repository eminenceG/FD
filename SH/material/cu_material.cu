#include"material.h"
#include"../param/param.h"
#include"../gpu.h"
#include<cstdio>

extern __constant__ float d_coef[5][4];

void MATERIAL::init_gpu_full(int deviceid,PARAM &param,MATERIAL &mat){
  nx=mat.nx;
  nz=mat.nz;
  int nz1=param.a_nz1[deviceid];
  int nz2=param.a_nz2[deviceid];
  int tnz=nz2-nz1+1;

  safecall(hipMemcpyToSymbol(HIP_SYMBOL(d_coef),g_coef,sizeof(float)*20));

  usetable=mat.usetable;
  if(usetable){
	num_mat=mat.num_mat;
	usetable=mat.usetable;
	safecall(hipMalloc((void**)&(tbl_BV ),sizeof(float)*num_mat));
	safecall(hipMalloc((void**)&(tbl_MUZ),sizeof(float)*num_mat));
	safecall(hipMalloc((void**)&(tbl_MUX),sizeof(float)*num_mat));
	safecall(hipMalloc((void**)&(index),sizeof(float)*nx*tnz)); index  -=nz1*nx;

	safecall(hipMemcpy( tbl_BV , mat.tbl_BV , sizeof(float)*num_mat,hipMemcpyHostToDevice));
	safecall(hipMemcpy( tbl_MUZ , mat.tbl_MUZ , sizeof(float)*num_mat,hipMemcpyHostToDevice));
	safecall(hipMemcpy( tbl_MUX, mat.tbl_MUX, sizeof(float)*num_mat,hipMemcpyHostToDevice));
	safecall(hipMemcpy( index  + nz1*nx, mat.index  + nz1*nx, sizeof(float)*nx*tnz,hipMemcpyHostToDevice));

  }else{
	safecall(hipMalloc((void**)&(BV ),sizeof(float)*nx*tnz)); BV  -=nz1*nx;
	safecall(hipMalloc((void**)&(MUX),sizeof(float)*nx*tnz));MUX  -=nz1*nx;
	safecall(hipMalloc((void**)&(MUZ),sizeof(float)*nx*tnz)); MUZ -=nz1*nx;

	safecall(hipMemcpy( BV  + nz1*nx, mat.BV  + nz1*nx, sizeof(float)*nx*tnz,hipMemcpyHostToDevice));
	safecall(hipMemcpy( MUX + nz1*nx, mat.MUX + nz1*nx, sizeof(float)*nx*tnz,hipMemcpyHostToDevice));
	safecall(hipMemcpy( MUZ + nz1*nx, mat.MUZ + nz1*nx, sizeof(float)*nx*tnz,hipMemcpyHostToDevice));

  }
}

